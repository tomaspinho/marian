
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hipblas.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hip/hip_fp16.h>
#include <chrono>

void GPU_fill_rand(half *A, int nr_rows_A, int nr_cols_A) {
     // Create a pseudo-random number generator
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

     // Set the seed for the random number generator using the system clock
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

     // Fill the array with random numbers on the device
     /* curandGenerateUniform(prng, A, nr_rows_A * nr_cols_A); */
}

void gpu_blas_mmul(const half *A, const half *B, half *C, const int m, const int k, const int n) {
     int lda=m,ldb=k,ldc=m;

     half alf_h;
     half *alpha_h = &alf_h;

     half bet_h;
     half *beta_h = &bet_h;

     // Create a handle for CUBLAS
     hipblasHandle_t handle;
     hipblasCreate(&handle);


     // Do the actual multiplication
     for (size_t i = 0; i < 1; ++i) {
        hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha_h,
            A, lda, B, ldb, beta_h, C, ldc);
     }
     hipStreamSynchronize(0);

     // Destroy the handle
     hipblasDestroy(handle);
}

int main() {
    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();

     // Allocate 3 arrays on CPU
     int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

     // for simplicity we are going to use square arrays
     nr_rows_A = 12;
     nr_cols_A = 500;
     nr_rows_B = 500;
     nr_cols_B = 90000;
     nr_rows_C = 12;
     nr_cols_C = 90000;

     // Allocate 3 arrays on GPU
     half *d_A, *d_B, *d_C;
     hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(half));
     hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(half));
     hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(half));

     for (size_t i = 0; i < 1000; ++i) {
		 // Fill the arrays A and B on GPU with random numbers
		 GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
		 GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

		 // Multiply A and B on GPU
		 gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
     }

     // Copy (and print) the result on host memory

     //Free GPU memory
     hipFree(d_A);
     hipFree(d_B);
     hipFree(d_C);  

     std::cerr << "COS\n";
     end = std::chrono::system_clock::now();

     std::chrono::duration<double> elapsed_seconds = end-start;
     std::time_t end_time = std::chrono::system_clock::to_time_t(end);
     std::cout << "finished computation at " << std::ctime(&end_time)
               << "elapsed time: " << elapsed_seconds.count() << "s\n";

     return 0;
 }
